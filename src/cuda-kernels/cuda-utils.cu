// BSD 3-Clause License; see https://github.com/jpivarski/awkward-1.0/blob/master/LICENSE

#include "awkward/kernels/cuda-utils.h"

ERROR awkward_cuda_ptr_device_num(int& device_num, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if (status != hipError_t::hipSuccess) {
    return failure(hipGetErrorString(status), 0, kSliceNone, true);
  }
  device_num = att.device;
  return success();
}

ERROR awkward_cuda_ptr_device_name(std::string& device_name, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if (status != hipError_t::hipSuccess) {
    return failure(hipGetErrorString(status), 0, kSliceNone, true);
  }

  hipDeviceProp_t dev_prop;
  status = hipGetDeviceProperties(&dev_prop, att.device);
  if (status != hipError_t::hipSuccess) {
    return failure(hipGetErrorString(status), 0, kSliceNone, true);
  }
  device_name = dev_prop.name;
  return success();
}

ERROR awkward_cuda_host_to_device(
  void* to_ptr,
  void* from_ptr,
  int64_t bytelength) {
  hipError_t memcpy_stat = hipMemcpy(to_ptr,
                                       from_ptr,
                                       bytelength,
                                       hipMemcpyHostToDevice);
  if (memcpy_stat != hipError_t::hipSuccess) {
    return failure(hipGetErrorString(memcpy_stat), kSliceNone, kSliceNone, true);
  }
  else {
    return success();
  }
}

ERROR awkward_cuda_device_to_host(
  void* to_ptr,
  void* from_ptr,
  int64_t bytelength) {
  hipError_t memcpy_stat = cudMemcpy(to_ptr,
                                      from_ptr,
                                      bytelength,
                                      hipMemcpyDeviceToHost);
  if (memcpy_stat != hipError_t::hipSuccess) {
    return failure(hipGetErrorString(memcpy_stat), kSliceNone, kSliceNone, true);
  }
  else {
    return success();
  }
}
