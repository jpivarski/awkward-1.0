#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T>
__global__ void
cuda_RegularArray_num(T* tonum, int64_t size, int64_t length, int64_t invocation_index, int64_t* err_code) {
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id < length && err_code[0] == MAX_NUMPY_INT) {
    tonum[thread_id] = size;
  }
}
