// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/master/LICENSE

#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/allocators.cu", line)

#include "awkward/kernels/allocators.h"

void* awkward_malloc(int64_t bytelength) {
  void* out = nullptr;
  if (bytelength != 0) {
    hipError_t err = hipMallocManaged(&out, bytelength);
    if (err != hipError_t::hipSuccess) {
      out = nullptr;
    }
  }
  return out;
}

void awkward_free(void const *ptr) {
  hipFree((void*)ptr);
}
