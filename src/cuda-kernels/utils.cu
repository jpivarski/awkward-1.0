#include "awkward/cuda-kernels/cuda_utils.h"

ERROR awkward_cuda_ptr_device_num(int& device_num, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if(status != hipError_t::hipSuccess)
    return failure(hipGetErrorString(status), 0, kSliceNone, true);
   device_num = att.device;
  return success();
}

ERROR awkward_cuda_ptr_device_name(std::string& device_name, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if(status != hipError_t::hipSuccess)
    return failure(hipGetErrorString(status), 0, kSliceNone, true);

  hipDeviceProp_t dev_prop;
  status = hipGetDeviceProperties(&dev_prop, att.device);
  if(status != hipError_t::hipSuccess)
    return failure(hipGetErrorString(status), 0, kSliceNone, true);
  device_name = dev_prop.name;
  return success();
}

