#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents,outlength, invocation_index,err_code) = args
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_max_complex_a", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents,outlength, invocation_index,err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_max_complex_b", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents,outlength, invocation_index,err_code))
// out["awkward_reduce_max_complex_a", {dtype_specializations}] = None
// out["awkward_reduce_max_complex_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_max_complex_a(T* toptr,
                     const C* fromptr,
                     const U* parents,
                     int64_t lenparents,
                     int64_t outlength,
                     T identity,
                     uint64_t invocation_index,
                     uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      toptr[thread_id * 2] = identity;
      toptr[thread_id * 2 + 1] = 0;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_max_complex_b(T* toptr,
                     const C* fromptr,
                     const U* parents,
                     int64_t lenparents,
                     int64_t outlength,
                     T identity,
                     uint64_t invocation_index,
                     uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < lenparents) {
      int64_t parent = parents[thread_id];
      C x = fromptr[thread_id * 2];
      C y = fromptr[thread_id * 2 + 1];
      if (x > toptr[parent * 2]  ||
        (x == toptr[parent * 2]  &&  y > toptr[parent * 2 + 1])) {
        toptr[parent * 2] = x;
        toptr[parent * 2 + 1] = y;
      }
    }
  }
}
