#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

typedef signed char int8_t;
typedef signed short int16_t;
typedef signed int int32_t;
typedef signed long long int64_t;
typedef signed char int_fast8_t;
typedef signed short int_fast16_t;
typedef signed int int_fast32_t;
typedef signed long long int_fast64_t;
typedef signed char int_least8_t;
typedef signed short int_least16_t;
typedef signed int int_least32_t;
typedef signed long long int_least64_t;
typedef signed long long intmax_t;
typedef signed long intptr_t;  //optional
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;
typedef unsigned char uint_fast8_t;
typedef unsigned short uint_fast16_t;
typedef unsigned int uint_fast32_t;
typedef unsigned long long uint_fast64_t;
typedef unsigned char uint_least8_t;
typedef unsigned short uint_least16_t;
typedef unsigned int uint_least32_t;
typedef unsigned long long uint_least64_t;
typedef unsigned long long uintmax_t;

#define RAISE_ERROR(ERROR_KERNEL_CODE) \
  atomicMin(err_code,                  \
            invocation_index*(1 << ERROR_BITS) + (int)(ERROR_KERNEL_CODE));

// BEGIN PYTHON
// def min_max_type(dtype):
//   supported_types = {
//       'bool': cupy.int32,
//       'int8': cupy.int32,
//       'int16': cupy.int32,
//       'int32': cupy.int32,
//       'int64': cupy.int64,
//       'uint8': cupy.uint32,
//       'uint16': cupy.uint32,
//       'uint32': cupy.uint32,
//       'uint64': cupy.uint64,
//       'float32': cupy.float32,
//       'float64': cupy.float64
//   }
//   if str(dtype) in supported_types:
//       return supported_types[str(dtype)]
//   else:
//       raise ValueError("Unsupported dtype.", dtype)
// END PYTHON

const int64_t  kMaxInt64  = 9223372036854775806;   // 2**63 - 2: see below
const int64_t  kSliceNone = kMaxInt64 + 1;         // for Slice::none()

void
awkward_regularize_rangeslice(
    int64_t* start,
    int64_t* stop,
    bool posstep,
    bool hasstart,
    bool hasstop,
    int64_t length) {
    if (posstep) {
      if (!hasstart)           *start = 0;
      else if (*start < 0)     *start += length;
      if (*start < 0)          *start = 0;
      if (*start > length)     *start = length;

      if (!hasstop)            *stop = length;
      else if (*stop < 0)      *stop += length;
      if (*stop < 0)           *stop = 0;
      if (*stop > length)      *stop = length;
      if (*stop < *start)      *stop = *start;
    }

    else {
      if (!hasstart)           *start = length - 1;
      else if (*start < 0)     *start += length;
      if (*start < -1)         *start = -1;
      if (*start > length - 1) *start = length - 1;

      if (!hasstop)            *stop = -1;
      else if (*stop < 0)      *stop += length;
      if (*stop < -1)          *stop = -1;
      if (*stop > length - 1)  *stop = length - 1;
      if (*stop > *start)      *stop = *start;
    }
  }
}

__device__ __forceinline__ float atomicMin(float* addr, float value) {
  float old; old = !signbit(value) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) : __uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));
  return old;
}
__device__ __forceinline__ float atomicMax(float* addr, float value) {
  float old; old = !signbit(value) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) : __uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));
  return old;
}

__device__ int64_t atomicAdd(int64_t* address, int64_t val) {
  uint64_t* address_as_ull = (uint64_t*)address;
  uint64_t old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, assumed + (uint64_t)val);
  } while (assumed != old);
  return (int64_t)old;
}
