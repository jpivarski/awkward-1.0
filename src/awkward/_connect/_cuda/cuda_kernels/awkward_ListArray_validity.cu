#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

enum Error {
    ERROR_START_STOP,    // message: "start[i] > stop[i]"
    ERROR_START_ZERO,    // message: "start[i] < 0"
    ERROR_STOP_CONTENT   // message: "stop[i] > len(content)"
};

template <typename C>
__global__ void
cuda_ListArray_validity(
  const C* starts,
  const C* stops,
  int64_t length,
  int64_t lencontent,
  int64_t invocation_index,
  int64_t* err_code) {
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id < length && err_code[0] == MAX_NUMPY_INT) {
    C start = starts[thread_id];
    C stop = stops[thread_id];
    if (start != stop) {
      if (start > stop) {
        atomicMin(err_code, invocation_index * (1 << ERROR_BITS) + ERROR_START_STOP); // failure("start[i] > stop[i]", i, kSliceNone, FILENAME(__LINE__));
      }
      if (start < 0) {
        atomicMin(err_code, invocation_index * (1 << ERROR_BITS) + ERROR_START_ZERO); // failure("start[i] < 0", i, kSliceNone, FILENAME(__LINE__));
      }
      if (stop > lencontent) {
        atomicMin(err_code, invocation_index * (1 << ERROR_BITS) + ERROR_STOP_CONTENT); // failure("stop[i] > len(content)", i, kSliceNone, FILENAME(__LINE__));
      }
    }
  }
}
