#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

__global__ void
awkward_RegularArray_localindex(int64_t* toindex,
                                int64_t size,
                                int64_t length,
                                uint64_t invocation_index,
                                uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      if (thready_dim < size) {
        toindex[((thread_id * size) + thready_dim)] = thready_dim;
      }
    }
  }
}
