#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

__global__ void
awkward_RegularArray_getitem_next_range_spreadadvanced(
    int64_t* toadvanced,
    const int64_t* fromadvanced,
    int64_t length,
    int64_t nextsize,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      if (thready_dim < nextsize) {
        toadvanced[((thread_id * nextsize) + thready_dim)] =
            fromadvanced[thread_id];
      }
    }
  }
}
