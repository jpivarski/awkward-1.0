#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, tocarry, mask, length, validwhen, invocation_index, err_code):
//     scan_in_array = cupy.empty(length, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(['awkward_ByteMaskedArray_getitem_nextcarry_a']))(grid, block, (mask, scan_in_array, mask, validwhen, length, invocation_index, err_code))
//     scan_in_array = inclusive_scan(grid, block, (scan_in_array, length, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(['awkward_ByteMaskedArray_getitem_nextcarry_b']))(grid, block, (scan_in_array, tocarry, mask, validwhen, length, invocation_index, err_code))
// END PYTHON

__global__ void
awkward_ByteMaskedArray_getitem_nextcarry_a(const int8_t* mask,
                                            int64_t* scan_in_array,
                                            bool validwhen,
                                            int64_t length,
                                            uint64_t* invocation_index,
                                            uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < length) {
      if ((mask[thread_id] != 0) == validwhen) {
        scan_in_array[thread_id] = 1;
      } else {
        scan_in_array[thread_id] = 0;
      }
    }
  }
}

__global__ void
awkward_ByteMaskedArray_getitem_nextcarry_b(int64_t* scan_in_array,
                                            int64_t* to_carry,
                                            const int8_t* mask,
                                            bool validwhen,
                                            int64_t length,
                                            uint64_t* invocation_index,
                                            uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < length) {
      if ((mask[thread_id] != 0) == validwhen) {
        to_carry[scan_in_array[thread_id] - 1] = thread_id;
      }
    }
  }
}
