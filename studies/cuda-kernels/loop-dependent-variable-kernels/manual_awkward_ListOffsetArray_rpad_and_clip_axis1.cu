#include "hip/hip_runtime.h"
#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/manual_awkward_ListOffsetArray_rpad_and_clip_axis1.cu", line)

#include "awkward/kernels/operations.h"
#include "standard_parallel_algorithms.h"

template <typename T, typename C>
__global__ void
awkward_ListOffsetArray_rpad_and_clip_axis1_kernel(
  T* toindex,
  const C* fromoffsets,
  int64_t length,
  int64_t target) {
  int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t thread_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(thread_idx < length) {
    int64_t rangeval = (T)(fromoffsets[thread_idx + 1] - fromoffsets[thread_idx]);
    int64_t shorter = (target < rangeval) ? target : rangeval;

	if(thread_idy < shorter) {
      toindex[thread_idx * target + thread_idy] = (T)fromoffsets[thread_idx] + thread_idy;
    }
	else if(thread_idy >= shorter && thread_idy < target) {
      toindex[thread_idx * target + thread_idy] = -1;
    }
  }
}

template <typename T, typename C>
ERROR awkward_ListOffsetArray_rpad_and_clip_axis1(
    T* toindex,
    const C* fromoffsets,
    int64_t length,
    int64_t target) {

  dim3 blocks_per_grid = blocks_2d(length, target);
  dim3 threads_per_block = threads_2d(length, target);

  awkward_ListOffsetArray_rpad_and_clip_axis1_kernel<T, C><<<blocks_per_grid, threads_per_block>>>(
      toindex,
      fromoffsets,
      length,
      target);

  hipDeviceSynchronize();

  return success();
}
ERROR awkward_ListOffsetArray32_rpad_and_clip_axis1_64(
    int64_t* toindex,
    const int32_t* fromoffsets,
    int64_t length,
    int64_t target) {
  return awkward_ListOffsetArray_rpad_and_clip_axis1<int64_t, int32_t>(
      toindex,
      fromoffsets,
      length,
      target);
}
ERROR awkward_ListOffsetArrayU32_rpad_and_clip_axis1_64(
    int64_t* toindex,
    const uint32_t* fromoffsets,
    int64_t length,
    int64_t target) {
  return awkward_ListOffsetArray_rpad_and_clip_axis1<int64_t, uint32_t>(
      toindex,
      fromoffsets,
      length,
      target);
}
ERROR awkward_ListOffsetArray64_rpad_and_clip_axis1_64(
    int64_t* toindex,
    const int64_t* fromoffsets,
    int64_t length,
    int64_t target) {
  return awkward_ListOffsetArray_rpad_and_clip_axis1<int64_t, int64_t>(
      toindex,
      fromoffsets,
      length,
      target);
}
