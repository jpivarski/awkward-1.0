#include "hip/hip_runtime.h"
#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/manual_awkward_IndexedArray_getitem_nextcarry_outindex_mask.cu", line)

#include "standard_parallel_algorithms.h"
#include "awkward/kernels.h"

template <typename C, typename T>
__global__
void awkward_IndexedArray_getitem_nextcarry_outindex_mask_kernel(
    T* tocarry,
    T* toindex,
    const C* fromindex,
    int64_t* prefixedsum_mask,
    int64_t lenindex,
    int64_t lencontent,
    unsigned long long* error_i) {

  /**
   * Here the thread_id has a unsigned long long data type rather than a int64_t
   * type because atomicMin doesn't provide a fucntion signature for int64_t type
   */
  unsigned long long thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if(thread_id < lenindex) {
    C j = fromindex[thread_id];

    if (j >= lencontent) {
      atomicMin(error_i, thread_id);
    } else if (j < 0) {
      toindex[thread_id] = -1;
    } else {
      tocarry[prefixedsum_mask[thread_id] - 1] = j;
      toindex[thread_id] = (T)(prefixedsum_mask[thread_id] - 1);
    }
  }
}

template <typename C>
__global__ void
awkward_IndexedArray_getitem_nextcarry_outindex_mask_filter_mask(
    const C* fromindex,
    int8_t* filtered_mask,
    int64_t lenindex,
    int64_t lencontent) {
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if(thread_id < lenindex) {
    if (fromindex[thread_id] < lencontent && fromindex[thread_id] >= 0) {
      filtered_mask[thread_id] = 1;
    }
  }
}

__global__ void
awkward_IndexedArray_getitem_nextcarry_outindex_mask_initialize_error_i(
    unsigned long long* error_i,
    unsigned long long value) {
  *error_i = value;
}

template <typename C, typename T>
ERROR awkward_IndexedArray_getitem_nextcarry_outindex_mask(
    T* tocarry,
    T* toindex,
    const C* fromindex,
    int64_t lenindex,
    int64_t lencontent) {

  dim3 blocks_per_grid = blocks(lenindex);
  dim3 threads_per_block = threads(lenindex);

  int8_t* filtered_mask;
  int64_t* res_temp;

  HANDLE_ERROR(hipMalloc((void**)&filtered_mask, sizeof(int8_t) * lenindex));
  HANDLE_ERROR(hipMalloc((void**)&res_temp, sizeof(int64_t) * lenindex));
  HANDLE_ERROR(hipMemset(filtered_mask, 0, sizeof(int8_t) * lenindex));

  awkward_IndexedArray_getitem_nextcarry_outindex_mask_filter_mask<C><<<blocks_per_grid, threads_per_block>>>(
      fromindex,
      filtered_mask,
      lenindex,
      lencontent);

  exclusive_scan(res_temp, filtered_mask, lenindex);

  unsigned long long * dev_error_i;
  unsigned long long error_i;

  HANDLE_ERROR(hipMalloc((void**)&dev_error_i, sizeof(unsigned long long)));
  awkward_IndexedArray_getitem_nextcarry_outindex_mask_initialize_error_i<<<1,1>>>(
      dev_error_i,
      lenindex + 1);

  awkward_IndexedArray_getitem_nextcarry_outindex_mask_kernel<C, T><<<blocks_per_grid, threads_per_block>>>(
      tocarry,
      toindex,
      fromindex,
      res_temp,
      lenindex,
      lencontent,
      dev_error_i);

  HANDLE_ERROR(hipMemcpy(&error_i, dev_error_i, sizeof(unsigned long long), hipMemcpyDeviceToHost));

  if(error_i != lenindex + 1) {
    C error_j;
    HANDLE_ERROR(hipMemcpy(&error_j, fromindex + error_i, sizeof(C), hipMemcpyDeviceToHost));
    return failure("index out of range", error_i, error_j, FILENAME(__LINE__));
  }

  return success();
}
ERROR awkward_IndexedArray32_getitem_nextcarry_outindex_mask_64(
    int64_t* tocarry,
    int64_t* toindex,
    const int32_t* fromindex,
    int64_t lenindex,
    int64_t lencontent) {
  return awkward_IndexedArray_getitem_nextcarry_outindex_mask<int32_t, int64_t>(
      tocarry,
      toindex,
      fromindex,
      lenindex,
      lencontent);
}
ERROR awkward_IndexedArrayU32_getitem_nextcarry_outindex_mask_64(
    int64_t* tocarry,
    int64_t* toindex,
    const uint32_t* fromindex,
    int64_t lenindex,
    int64_t lencontent) {
  return awkward_IndexedArray_getitem_nextcarry_outindex_mask<uint32_t, int64_t>(
      tocarry,
      toindex,
      fromindex,
      lenindex,
      lencontent);
}
ERROR awkward_IndexedArray64_getitem_nextcarry_outindex_mask_64(
    int64_t* tocarry,
    int64_t* toindex,
    const int64_t* fromindex,
    int64_t lenindex,
    int64_t lencontent) {
  return awkward_IndexedArray_getitem_nextcarry_outindex_mask<int64_t, int64_t>(
      tocarry,
      toindex,
      fromindex,
      lenindex,
      lencontent);
}

