#include "hip/hip_runtime.h"
#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/manual_awkward_ListOffsetArray_rpad_axis1.cu", line)

#include "awkward/kernels/operations.h"
#include "standard_parallel_algorithms.h"

template <typename T, typename C>
__global__
void awkward_ListOffsetArray_rpad_axis1_kernel(
  T* toindex,
  const C* fromoffsets,
  int64_t fromlength,
  int64_t target) {
  int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t thread_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(thread_idx < fromlength) {
    int64_t rangeval = (T)(fromoffsets[thread_idx + 1] - fromoffsets[thread_idx]);

	  if(thread_idy < rangeval) {
      toindex[thread_idx * target + thread_idy] = (T)fromoffsets[thread_idx] + thread_idy;
    }
	  else if(thread_idy >= rangeval && thread_idy < target) {
      toindex[thread_idx * target + thread_idy] = -1;
    }
  }
}
    
template <typename T, typename C>
ERROR awkward_ListOffsetArray_rpad_axis1(
  T* toindex,
  const C* fromoffsets,
  int64_t fromlength,
  int64_t target) {
  dim3 blocks_per_grid = blocks_2d(fromlength, target);
  dim3 threads_per_block = threads_2d(fromlength, target);

  awkward_ListOffsetArray_rpad_axis1_kernel<T, C><<<blocks_per_grid, threads_per_block>>>(
      toindex,
      fromoffsets,
      fromlength,
      target);

  hipDeviceSynchronize();
  return success();
}
ERROR awkward_ListOffsetArray32_rpad_axis1_64(
  int64_t* toindex,
  const int32_t* fromoffsets,
  int64_t fromlength,
  int64_t target) {
  return awkward_ListOffsetArray_rpad_axis1<int64_t, int32_t>(
    toindex,
    fromoffsets,
    fromlength,
    target);
}
ERROR awkward_ListOffsetArrayU32_rpad_axis1_64(
  int64_t* toindex,
  const uint32_t* fromoffsets,
  int64_t fromlength,
  int64_t target) {
  return awkward_ListOffsetArray_rpad_axis1<int64_t, uint32_t>(
    toindex,
    fromoffsets,
    fromlength,
    target);
}
ERROR awkward_ListOffsetArray64_rpad_axis1_64(
  int64_t* toindex,
  const int64_t* fromoffsets,
  int64_t fromlength,
  int64_t target) {
  return awkward_ListOffsetArray_rpad_axis1<int64_t, int64_t>(
    toindex,
    fromoffsets,
    fromlength,
    target);
}
