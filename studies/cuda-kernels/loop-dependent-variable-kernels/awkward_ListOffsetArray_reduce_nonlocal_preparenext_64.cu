#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/master/LICENSE

#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/awkward_ListOffsetArray_reduce_nonlocal_preparenext_64.cpp", line)

#include "standard_parallel_algorithms.h"
#include "awkward/kernels.h"

__global__
void awkward_ListOffsetArray_reduce_nonlocal_preparenext_64_initialize_distincts(
  int64_t* distincts,
  int64_t distinctlen) {

  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;;

  if(thread_id < distinctlen) {
    distincts[thread_id] = -1;
  }
}
ERROR awkward_ListOffsetArray_reduce_nonlocal_preparenext_64(
  int64_t* nextcarry,
  int64_t* nextparents,
  int64_t nextlen,
  int64_t* maxnextparents,
  int64_t* distincts,
  int64_t distinctslen,
  int64_t* offsetscopy,
  const int64_t* offsets,
  int64_t length,
  const int64_t* parents,
  int64_t maxcount) {
  *maxnextparents = 0;

  dim3 blocks_per_grid = blocks(distinctslen);
  dim3 threads_per_block = threads(distinctslen);

  awkward_ListOffsetArray_reduce_nonlocal_preparenext_64_initialize_distincts<<<blocks_per_grid, threads_per_block>>>(
    distincts,
    distinctslen);

  blocks_per_grid = blocks(nextlen);
  threads_per_block = threads(nextlen);

  int8_t* k_mask_arr;

  HANDLE_ERROR(hipMalloc(k_mask_arr, sizeof(int8_t) * length));
  HANDLE_ERROR(hipMemset(k_mask_arr, nextlen, 0));
  awkward_ListOffsetArray_reduce_nonlocal_preparenext_64_k_mask(
      k_mask_arr)
  int64_t k = 0;
  while (k < nextlen) {
    int64_t j = 0;
    for (int64_t i = 0;  i < length;  i++) {
      if (offsetscopy[i] < offsets[i + 1]) {
        int64_t diff = offsetscopy[i] - offsets[i];
        int64_t parent = parents[i];

        nextcarry[k] = offsetscopy[i];
        nextparents[k] = parent*maxcount + diff;

        if (*maxnextparents < nextparents[k]) {
          *maxnextparents = nextparents[k];
        }

        if (distincts[nextparents[k]] == -1) {
          distincts[nextparents[k]] = j;
          j++;
        }

        k++;
        offsetscopy[i]++;
      } 
    }
  }
  return success();
}
