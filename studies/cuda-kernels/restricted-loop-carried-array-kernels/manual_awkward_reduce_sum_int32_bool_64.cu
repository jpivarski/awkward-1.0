#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

#define FILENAME(line)          \
  FILENAME_FOR_EXCEPTIONS_CUDA( \
      "src/cuda-kernels/awkward_reduce_sum_int32_bool_64.cu", line)

#include "standard_parallel_algorithms.h"
#include "awkward/kernels.h"

__global__ void
awkward_reduce_sum_int32_bool_64_kernel(int32_t* toptr,
                                        const bool* fromptr,
                                        const int64_t* parents,
                                        int64_t lenparents) {
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id < lenparents) {
    toptr[parents[thread_id]] += (fromptr[thread_id] != 0);
  }
}

ERROR
awkward_reduce_sum_int32_bool_64(int32_t* toptr,
                                 const bool* fromptr,
                                 const int64_t* parents,
                                 int64_t lenparents,
                                 int64_t outlength) {
  HANDLE_ERROR(hipMemset(toptr, 0, sizeof(int32_t) * outlength));

  dim3 blocks_per_grid = blocks(lenparents);
  dim3 threads_per_block = threads(lenparents);

  awkward_reduce_sum_int32_bool_64_kernel<<<blocks_per_grid, threads_per_block>>>(
      toptr, fromptr, parents, lenparents);

  return success();
}
