#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/master/LICENSE

#define FILENAME(line) \
  FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/awkward_reduce_prod.cu", line)

#include "standard_parallel_algorithms.h"
#include "awkward/kernels.h"

template <typename OUT, typename IN>
__global__ void
awkward_reduce_prod_kernel(OUT* toptr,
                          const IN* fromptr,
                          const int64_t* parents,
                          int64_t lenparents) {
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id < lenparents) {
    toptr[parents[thread_id]] *= (OUT)fromptr[thread_id];
  }
}

template <typename OUT, typename IN>
ERROR
awkward_reduce_prod(OUT* toptr,
                   const IN* fromptr,
                   const int64_t* parents,
                   int64_t lenparents,
                   int64_t outlength) {
  HANDLE_ERROR(hipMemset(toptr, 1, sizeof(OUT) * outlength));

  dim3 blocks_per_grid = blocks(lenparents);
  dim3 threads_per_block = threads(lenparents);

  awkward_reduce_prod_kernel<<<blocks_per_grid, threads_per_block>>>(
      toptr, fromptr, parents, lenparents);

  return success();
}
ERROR
awkward_reduce_prod_int64_int8_64(int64_t* toptr,
                                 const int8_t* fromptr,
                                 const int64_t* parents,
                                 int64_t lenparents,
                                 int64_t outlength) {
  return awkward_reduce_prod<int64_t, int8_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint64_uint8_64(uint64_t* toptr,
                                   const uint8_t* fromptr,
                                   const int64_t* parents,
                                   int64_t lenparents,
                                   int64_t outlength) {
  return awkward_reduce_prod<uint64_t, uint8_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_int64_int16_64(int64_t* toptr,
                                  const int16_t* fromptr,
                                  const int64_t* parents,
                                  int64_t lenparents,
                                  int64_t outlength) {
  return awkward_reduce_prod<int64_t, int16_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint64_uint16_64(uint64_t* toptr,
                                    const uint16_t* fromptr,
                                    const int64_t* parents,
                                    int64_t lenparents,
                                    int64_t outlength) {
  return awkward_reduce_prod<uint64_t, uint16_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_int64_int32_64(int64_t* toptr,
                                  const int32_t* fromptr,
                                  const int64_t* parents,
                                  int64_t lenparents,
                                  int64_t outlength) {
  return awkward_reduce_prod<int64_t, int32_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint64_uint32_64(uint64_t* toptr,
                                    const uint32_t* fromptr,
                                    const int64_t* parents,
                                    int64_t lenparents,
                                    int64_t outlength) {
  return awkward_reduce_prod<uint64_t, uint32_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_int64_int64_64(int64_t* toptr,
                                  const int64_t* fromptr,
                                  const int64_t* parents,
                                  int64_t lenparents,
                                  int64_t outlength) {
  return awkward_reduce_prod<int64_t, int64_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint64_uint64_64(uint64_t* toptr,
                                    const uint64_t* fromptr,
                                    const int64_t* parents,
                                    int64_t lenparents,
                                    int64_t outlength) {
  return awkward_reduce_prod<uint64_t, uint64_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_float32_float32_64(float* toptr,
                                      const float* fromptr,
                                      const int64_t* parents,
                                      int64_t lenparents,
                                      int64_t outlength) {
  return awkward_reduce_prod<float, float>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_float64_float64_64(double* toptr,
                                      const double* fromptr,
                                      const int64_t* parents,
                                      int64_t lenparents,
                                      int64_t outlength) {
  return awkward_reduce_prod<double, double>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_int32_int8_64(int32_t* toptr,
                                 const int8_t* fromptr,
                                 const int64_t* parents,
                                 int64_t lenparents,
                                 int64_t outlength) {
  return awkward_reduce_prod<int32_t, int8_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint32_uint8_64(uint32_t* toptr,
                                   const uint8_t* fromptr,
                                   const int64_t* parents,
                                   int64_t lenparents,
                                   int64_t outlength) {
  return awkward_reduce_prod<uint32_t, uint8_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_int32_int16_64(int32_t* toptr,
                                  const int16_t* fromptr,
                                  const int64_t* parents,
                                  int64_t lenparents,
                                  int64_t outlength) {
  return awkward_reduce_prod<int32_t, int16_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint32_uint16_64(uint32_t* toptr,
                                    const uint16_t* fromptr,
                                    const int64_t* parents,
                                    int64_t lenparents,
                                    int64_t outlength) {
  return awkward_reduce_prod<uint32_t, uint16_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_int32_int32_64(int32_t* toptr,
                                  const int32_t* fromptr,
                                  const int64_t* parents,
                                  int64_t lenparents,
                                  int64_t outlength) {
  return awkward_reduce_prod<int32_t, int32_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
ERROR
awkward_reduce_prod_uint32_uint32_64(uint32_t* toptr,
                                    const uint32_t* fromptr,
                                    const int64_t* parents,
                                    int64_t lenparents,
                                    int64_t outlength) {
  return awkward_reduce_prod<uint32_t, uint32_t>(
      toptr, fromptr, parents, lenparents, outlength);
}
